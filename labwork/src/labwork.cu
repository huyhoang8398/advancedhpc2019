#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            printf("labwork 1 CPU OpenMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            printf("labwork 3 elapsed %.1fms\n", timer.getElapsedTimeInMilliSec());
            break;
        //case 4:
        //    labwork.labwork4_GPU();
        //    labwork.saveOutputImage("labwork4-gpu-out.jpg");
        //    break;
        //case 5:
        //    labwork.labwork5_CPU();
        //    labwork.saveOutputImage("labwork5-cpu-out.jpg");
        //    labwork.labwork5_GPU();
        //    labwork.saveOutputImage("labwork5-gpu-out.jpg");
        //    break;
        //case 6:
        //    labwork.labwork6_GPU();
        //    labwork.saveOutputImage("labwork6-gpu-out.jpg");
        //    break;
        //case 7:
        //    labwork.labwork7_GPU();
        //    printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        //    labwork.saveOutputImage("labwork7-gpu-out.jpg");
        //    break;
        //case 8:
        //    labwork.labwork8_GPU();
        //    printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        //    labwork.saveOutputImage("labwork8-gpu-out.jpg");
        //    break;
        //case 9:
        //    labwork.labwork9_GPU();
        //    printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        //    labwork.saveOutputImage("labwork9-gpu-out.jpg");
        //    break;
        //case 10:
        //    labwork.labwork10_GPU();
        //    printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        //    labwork.saveOutputImage("labwork10-gpu-out.jpg");
        //    break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    // do something here
    omp_set_num_threads(4);
    #pragma omp parallel
    printf("%d\n", omp_get_num_threads());
    for (int i = 0; i < pixelCount; i++) {
      outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
				    (int) inputImage->buffer[i * 3 + 2]) / 3);
      outputImage[i * 3 + 1] = outputImage[i * 3];
      outputImage[i * 3 + 2] = outputImage[i * 3];
    }
    
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int nDevices;
    if (hipGetDeviceCount(&nDevices) != hipSuccess) {
        fprintf(stderr, "cannot get number of devices\n");
        return;
    }
    printf("%d devices found\n", nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) != hipSuccess) {
            fprintf(stderr, "cannot get device props\n");
            return;
        }
        printf("Information for device %d:\n", i);
        printf("Device name: %s\n", prop.name);
        int cores = getSPcores(prop);
        printf("Core count: %d\n", cores);
        printf("Core clock rate: %d kHz\n", prop.clockRate);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Warp size: %d threads\n", prop.warpSize);
        printf("Memory clock rate: %d kHz\n", prop.memoryClockRate);
        printf("Memory bus width: %d bits\n", prop.memoryBusWidth);
        printf("\n");
}

}
//
void Labwork::labwork3_GPU() {
    // Calculate number of pixels
    long long pixelCount = inputImage->width * inputImage->height;
    char *blockSizeEnv = getenv("LW3_CUDA_BLOCK_SIZE");
    if (!blockSizeEnv) {
        fprintf(stderr, "invalid block size\n");
        return 0;
    }

    // Allocate CUDA memory    
    int blockSize = atoi(blockSizeEnv);
    long long numBlocks = pixelCount / blockSize + 1;

    // Copy CUDA Memory from CPU to GPU
    // Processing
    // Copy CUDA Memory from GPU to CPU
    uchar3 *inputCudaBuffer;
    if (hipMalloc(&inputCudaBuffer, pixelCount * sizeof(uchar3)) != hipSuccess) {
        fprintf(stderr, "memory allocation error\n");
        return 0;
    }
    uchar3 *outputCudaBuffer;
    if (hipMalloc(&outputCudaBuffer, pixelCount * sizeof(uchar3)) != hipSuccess) {
        fprintf(stderr, "memory allocation error\n");
        return 0;
    }

    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    if (hipMemcpy(inputCudaBuffer, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "input buffer copy error\n");
        return 0;
    }
    for (int j = 0; j < 100; j++) {
        labwork3<<<numBlocks, blockSize>>>(inputCudaBuffer, outputCudaBuffer, pixelCount);
    }
    if (hipMemcpy(outputImage, outputCudaBuffer, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "output buffer copy error\n");
        return 0;
    }
    // Cleaning
    hipFree(inputCudaBuffer);
    hipFree(outputCudaBuffer);

    return 1;
}

__global__ void labwork3(uchar3 * __restrict__ input, uchar3 * __restrict__ output, long long pixelCount) {
    long long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < pixelCount) {
        output[i].x = (char)(((int)input[i].x + input[i].y + input[i].z) / 3);
        output[i].y = output[i].z = output[i].x;
    }
}
//
//void Labwork::labwork4_GPU() {
//}
//
//void Labwork::labwork5_GPU(bool shared) {
//}
//
//void Labwork::labwork6_GPU() {
//}
//
//void Labwork::labwork7_GPU() {
//}
//
//void Labwork::labwork8_GPU() {
//}
//
//void Labwork::labwork9_GPU() {
//
//}
//
//void Labwork::labwork10_GPU(){
//}
//

























